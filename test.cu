#include "hip/hip_runtime.h"
#include<time.h>
#include<stdlib.h>
#include<math.h>
#include<valarray>
using namespace std;

__global__ void pi(int seed, float *estimate) {

hiprandState_t state;
int tid = threadIdx.x + blockIdx.x*blockDim.x;
int points =0;
float x,y;
// 4000/6000 trials per thread 
hiprand_init(seed,tid,0,&state);
for (int i=0;i<6000;i++)
{
x = hiprand_uniform(&state);
y = hiprand_uniform(&state);   
if(x*x + y*y <=1.0) 
points++;
}
estimate[tid]=points*4.0/6000;
}

int main() {
  int blocks =300;
  int threads =300; 
  float estimate[threads*blocks];  
  float *dev_est;
  hipMalloc( (void**) &dev_est,threads*blocks*sizeof(float));
  pi<<<threads,blocks>>>(time(NULL),dev_est);
  hipMemcpy(estimate, dev_est, threads*blocks*sizeof(float), hipMemcpyDeviceToHost);

  valarray<float> myvalarray(estimate,threads*blocks);
  float pi = myvalarray.sum()/float(threads*blocks);
  cout<<pi;
  return 0;
}

