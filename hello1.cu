#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<map> 
#include<vector> 
using namespace std;


__global__ void add(int *a, int *b) {

map <double, vector<double> > row_list; 
int tid = blockIdx.x; 
row_list[tid].push_back(a[tid]);
row_list[tid].push_back(b[tid]); 
 

}



int main
(void) { 

int a[1000], b[1000]; 
int *dev_a, *dev_b; 
hipMalloc( (void**)&dev_a, sizeof(a));
hipMalloc( (void**)&dev_b, sizeof(b)); 

for (int i=0; i<1000; i++) 
{

 a[i] = i; 
 b[i] = i*i; 

} 

hipMemcpy(dev_a, a, sizeof(a), hipMemcpyHostToDevice); 
hipMemcpy(dev_b, b, sizeof(b), hipMemcpyHostToDevice); 

//add<<<1000,1>>>(dev_a, dev_b); 

return 0; 

}
