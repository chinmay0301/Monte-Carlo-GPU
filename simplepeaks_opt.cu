#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h> 
#include<hiprand/hiprand_kernel.h> 
#include<hiprand.h>
#include<stdio.h>
#include<hip/hip_runtime.h> 
#include<Eigen/Eigen> 
#include<iostream> 
#include"math.h"
#include<time.h>  
#include<algorithm> 
#include<thrust/sort.h> 
using namespace std; 
using namespace Eigen; 


__device__ double S(float *x) {

double out = 3*pow((double)(1-x[0]),2)*exp(-pow((double)x[0],2) - pow((double)(x[1]+1),2)) - 10*(x[0]/5 - pow((double)x[0],3) - pow((double)x[1],5))*exp((double)-pow((double)x[0],2) - pow((double)x[1],2)) - 1/3*exp((double)-pow((double)(x[0]+1),2) -pow((double)x[1],2)); 

return out; 

} 

__global__ void math_tester(int seed, double *mu, double*sigma, double *x_0, double *x_1, double *sample) {

int tid = threadIdx.x + blockIdx.x*blockDim.x; 

double x0_ker, x1_ker, s_max;
double x0_max, x1_max; 
s_max = -500000; 

hiprandState_t state; 
hiprand_init(seed, tid, 0, &state); 
for (int i=0; i<320; i++)
{
 //hiprand_init(seed, tid, 0, &state); 
 x0_ker = hiprand_normal(&state)*sigma[0] + mu[0];
 x1_ker = hiprand_normal(&state)*sigma[1] + mu[1]; 
 float x[2] = {x0_ker, x1_ker};
 if(s_max < S(x)) 
 {
    s_max = S(x); 
    x0_max = x0_ker;
    x1_max = x1_ker; 
 }

}



//if (tid < 10240)
//{

//hiprand_init(seed, tid, 0, &state); 
//x_0[tid] = hiprand_normal(&state)*sigma[0] + mu[0]; 
//x_1[tid] = hiprand_normal(&state)*sigma[1] + mu[1];
//float x[2] = {x_0[tid], x_1[tid]}; 
sample[tid] = s_max;
x_0[tid] = x0_max;
x_1[tid] = x1_max;
//tid+= blockDim.x + gridDim.x;
//} 
__syncthreads();

}

int main () 

{


 int threads = 100;
 int blocks = 100;
double *dev_mu, *dev_sigma, *dev_x0, *dev_x1, *dev_sample; 

double mu[2]= {-3,-3};
double sigma[2] = {3,3};
double x0[threads*blocks];
double x1[threads*blocks]; 
double sample_0[threads*blocks]; 
double sample_1[threads*blocks]; 

hipMalloc( (void**)&dev_mu, 2*sizeof(double)); 
hipMalloc( (void**)&dev_sigma, 2*sizeof(double)); 
hipMalloc( (void**)&dev_x0, threads*blocks*sizeof(double));
hipMalloc( (void**)&dev_x1, threads*blocks*sizeof(double)); 
hipMalloc( (void**)&dev_sample, threads*blocks*sizeof(double));

while ( max(sigma[0], sigma[1]) > 0.00001) 
{  

hipMemcpy(dev_mu, &mu, 2*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(dev_sigma, &sigma, 2*sizeof(double), hipMemcpyHostToDevice);

math_tester<<<blocks,threads>>>(time(NULL), dev_mu, dev_sigma, dev_x0, dev_x1, dev_sample); 

hipMemcpy(&x0, dev_x0, threads*blocks*sizeof(double), hipMemcpyDeviceToHost); 
hipMemcpy(&x1, dev_x1, threads*blocks*sizeof(double), hipMemcpyDeviceToHost);
hipMemcpy(&sample_0, dev_sample, threads*blocks*sizeof(double), hipMemcpyDeviceToHost); 
hipMemcpy(&sample_1, dev_sample, threads*blocks*sizeof(double), hipMemcpyDeviceToHost); 

double mu0=0, mu1=0, sigma0=0, sigma1=0; 

thrust::sort_by_key(sample_0, sample_0 + threads*blocks, x0);
thrust::sort_by_key(sample_1, sample_1 + threads*blocks, x1); 
for (int i=threads*blocks*0.9; i<threads*blocks; i++)
{
   mu0 += x0[i];
   mu1 += x1[i];
}

mu[0] = mu0/double((0.1*threads*blocks));
mu[1] = mu1/double((0.1*threads*blocks));
 

for ( int i =threads*blocks*0.9; i<threads*blocks; i++) 
 {
	 sigma0 += pow((x0[i]-mu[0]),2);
	 sigma1 += pow((x1[i]-mu[1]),2); 
 }
sigma[0] = sqrt(sigma0/double((0.1*threads*blocks))); 
sigma[1] = sqrt(sigma1/double((0.1*threads*blocks)));

cout<<sample_0[9]<<" "<<sigma[0]<<" "<<sigma[1]<<" "<<mu[0]<<" "<<mu[1]<<"\n"; 
}
return 0;
}










