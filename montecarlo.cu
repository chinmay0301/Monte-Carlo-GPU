#include<iostream>
#include<stdio.h>
#include<hip/hip_runtime.h>

#include<stdlib.h>
#include<time.h>
#include<math.h>
#include<ctime>
#include<hiprand/hiprand.h> 
#include<hiprand/hiprand_kernel.h> 

using namespace std;

__device__ long int n_ok[1]={3};

__global__ void pi (int seed, float *x, float *y) { 

hiprandState_t state;
int tid = threadIdx.x + blockIdx.x*blockDim.x; 
if(tid < 1000)
{
hiprand_init(seed,tid,0,&state); 
*x = hiprand_uniform(&state);
*y = hiprand_uniform(&state); 
if ((*x)*(*x) + (*y)*(*y)>=1)
n_ok[0]++;
}
}
int main () {

/*long int n=1000000000;
long int n_ok =0;
float x,y;

srand(time(0));

for (long int i=0;i<n;i++)
{
x = rand();
x = x/RAND_MAX;
//srand(time(0)); 
y = rand();
y = y/RAND_MAX;

if(sqrt(x*x + y*y) <=1)
n_ok++;
}
float pi = 4.0 * n_ok/n;
cout<<"pi is roughly " <<pi; */ 
long int * n_ok1 = new long int;
float *x = new float;
float *y = new float; 
float *dev_x, *dev_y;  
hipMalloc( (void**)&dev_x, sizeof(float));
hipMalloc( (void**)&dev_y, sizeof(float));
//cudaMalloc( (void**)&dev_n, sizeof(long int));    
pi<<<128,128>>>(time(NULL), dev_x, dev_y); 
hipMemcpy(x, dev_x, sizeof(float), hipMemcpyDeviceToHost); 
hipMemcpy(y, dev_y, sizeof(float), hipMemcpyDeviceToHost);
hipMemcpyFromSymbol(n_ok1, HIP_SYMBOL("n_ok"), sizeof(long int), hipMemcpyDeviceToHost); 
cout<< *x<<" "<<*y<<" "<<*n_ok1 ; 
//cudaFree(dev_n); 
hipFree(dev_x);
hipFree(dev_y);
hipFree(n_ok1);
return 0;
}                      
