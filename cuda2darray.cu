#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<>
#include<iostream>

using namespace std; 
//#define BLOCKSIZE_x 16
//#define BLOCKSIZE_y 16

//#define Nrows 3
//#define Ncols 5

/*****************/
/* CUDA MEMCHECK */
/*****************/
//#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

/*inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
      fprintf(stderr, "GPUassert: %s %s %dn", hipGetErrorString(code), file, line);
      if (abort) {  exit(code); }
    }
}
*/
/*******************/
/* iDivUp FUNCTION */
/*******************/
//int iDivUp(int hostPtr, int b){ return ((hostPtr % b) != 0) ? (hostPtr / b + 1) : (hostPtr / b); }

/******************/
/* TEST KERNEL 2D */
/******************/
/*__global__ void test_kernel_2D(float *devPtr, size_t pitch)
{
   int tidx = blockIdx.x*blockDim.x + threadIdx.x;
   int tidy = blockIdx.y*blockDim.y + threadIdx.y;
   int arr[5] = {10,20,30,40,50}; 
   if ((tidx < Ncols) && (tidy < Nrows))
   {
       float *row_a = (float *)((char*)devPtr + tidy * pitch);
       row_a[tidx] = arr[tidx];
    }
}
*/
/********/
/* MAIN */
/********/
/*int main()
{
   float hostPtr[Nrows][Ncols];
   float *devPtr;
   size_t pitch;

   for (int i = 0; i < Nrows; i++)
   for (int j = 0; j < Ncols; j++) {
   hostPtr[i][j] = 1.f;
   //printf("row %i column %i value %f \n", i, j, hostPtr[i][j]);
}

// --- 2D pitched allocation and host->device memcopy
gpuErrchk(hipMallocPitch(&devPtr, &pitch, Ncols * sizeof(float), Nrows));
gpuErrchk(hipMemcpy2D(devPtr, pitch, hostPtr, Ncols*sizeof(float), Ncols*sizeof(float), Nrows, hipMemcpyHostToDevice));

dim3 gridSize(iDivUp(Ncols, BLOCKSIZE_x), iDivUp(Nrows, BLOCKSIZE_y));
dim3 blockSize(BLOCKSIZE_y, BLOCKSIZE_x);

test_kernel_2D << <gridSize, blockSize >> >(devPtr, pitch);
gpuErrchk(hipPeekAtLastError());
gpuErrchk(hipDeviceSynchronize());

gpuErrchk(hipMemcpy2D(hostPtr, Ncols * sizeof(float), devPtr, pitch, Ncols * sizeof(float), Nrows, hipMemcpyDeviceToHost));

for (int i = 0; i < Nrows; i++)
   for (int j = 0; j < Ncols; j++)
      printf("row %i column %i value %f \n", i, j, hostPtr[i][j]);

return 0;

}*/ 

#define BLOCK_SIZE 16
#define GRID_SIZE 1
__global__ void YourKernel(int d_A[BLOCK_SIZE][BLOCK_SIZE], int d_B[BLOCK_SIZE][BLOCK_SIZE]){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
//	if (row >= h || col >= w)return;
	d_B[row][col] = row-col; 
	d_A[row][col] = row+col; 
	/* whatever you wanna do with d_A[][] and d_B[][] */
}


int main(){
	int *d_A[BLOCK_SIZE][BLOCK_SIZE];
	int *d_B[BLOCK_SIZE][BLOCK_SIZE];

	int a[16][16];
	int b[16][16];
	/* d_A initialization */
        hipMalloc( (void**)&d_A, 256*sizeof(int));
	hipMalloc( (void**)&d_B, 256*sizeof(int));

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); // so your threads are BLOCK_SIZE*BLOCK_SIZE, 256 in this case
	dim3 dimGrid(GRID_SIZE, GRID_SIZE); // 1*1 blocks in a grid
         
	YourKernel<<<dimGrid, dimBlock>>>(d_A,d_B); //Kernel invocation
        hipMemcpy(a, d_A, 256*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(b, d_B, 256*sizeof(int), hipMemcpyDeviceToHost); 
         
	 
	for(int i=0;i<16;i++)
		for(int j=0; j<16;j++)
			cout<<a[i][j]<<" "; 
        return 0; 
}
