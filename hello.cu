#include<iostream>
#include<hip/hip_runtime.h>

#include<hiprand/hiprand.h>

using namespace std; 

__global__ void add(int*a, int*b, int*c) {

int tid = blockIdx.x;
 if(tid<1000) 
c[tid] = a[tid] + b[tid]; 

} 

int main(void) {

int a[1000], b[1000], c[1000]; 
int *dev_a, *dev_b, *dev_c;
hipMalloc( (void**)&dev_a, 1000*sizeof(int));
hipMalloc( (void**)&dev_b, 1000*sizeof(int));
hipMalloc( (void**)&dev_c, 1000*sizeof(int)); 

for ( int i=0; i<1000; i++) 
{

 a[i] =i;
b[i] = i*i; 

} 

hipMemcpy(dev_a, a, 1000*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(dev_b, b, 1000*sizeof(int), hipMemcpyHostToDevice); 
add<<<1000,1>>>(dev_a, dev_b, dev_c); 
hipMemcpy(c, dev_c, 1000*sizeof(int), hipMemcpyDeviceToHost); 

for (int i=0; i<1000; i++) 
{
cout<<c[i]<<"\n"; 
}
return 0; 
}
