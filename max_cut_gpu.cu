#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<hiprand/hiprand_kernel.h>
#include<hiprand.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<Eigen/Eigen>
#include<iostream>
#include<time.h>
#include"math.h"
#include<algorithm>
#include<thrust/sort.h> 

using namespace std;
using namespace Eigen; 



__device__ double m_data[36] = {0,0,200,0,0,0,609,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
//__constant__ double m_data[36]; 

__device__ double S(int *arr) 

{
	double out =0; 
	for (int i=0; i<6; i++)
	{
		for(int j=0; j<=5-i; j++)
		{
			if(arr[i]!=arr[j])
				out += m_data[6*j +i]; 
		}
	}
		return out; 
}


__global__ void max_cut(int seed, double *p_arr, int *s, int *n1, int *n2, int *n3, int *n4, int *n5, int *n6) // n1,n2 etc are arrays of 1s, and 0s 
{

hiprandState_t state; 
int tid = threadIdx.x + blockIdx.x*blockDim.x;
hiprand_init(seed,tid,0,&state); 
int a[6], a_max[6];
int s_max=0; 
for(int i=0; i<100; i++) 
{
 a[0] = (hiprand_uniform(&state)<p_arr[0]) ? 1 : 0;
 a[1] = (hiprand_uniform(&state)<p_arr[1]) ? 1 : 0;
 a[2] = (hiprand_uniform(&state)<p_arr[2]) ? 1 : 0; 
 a[3] = (hiprand_uniform(&state)<p_arr[3]) ? 1 : 0; 
 a[4] = (hiprand_uniform(&state)<p_arr[4]) ? 1 : 0;
 a[5] = (hiprand_uniform(&state)<p_arr[5]) ? 1 : 0;


if(s_max<S(a))
{
	s_max = S(a); 
        for (int j=0; j<6; j++) 
		a_max[j] = a[j]; 
}

}
s[tid] = s_max;
n1[tid] = a_max[0];
n2[tid] = a_max[1];
n3[tid] = a_max[2];
n4[tid] = a_max[3];
n5[tid] = a_max[4];
n6[tid] = a_max[5];

__syncthreads(); 
}


int main()
{

	double arr_zero[6] = {0,0,0,0,0,0};
        MatrixXd m = MatrixXd::Zero(6,6); 
        
	int threads = 250;
	int blocks = 10; 
	m(0,1) = 100;
	m(2,3) = 150;
	m(4,1) = 20; 
        double err;    
	double *m_host_data = m.data();
	//double host_data[36]; 
	
//	for(int i=0; i<36; i++) 
   //        host_data[i] = m_host_data[i];       
//        double *m_test_data; 
       
  //   hipMemcpyToSymbol(HIP_SYMBOL("m_data"), &host_data[0], sizeof(host_data), size_t(0), hipMemcpyHostToDevice);
   //    hipMemcpyFromSymbol(&m_test_data, HIP_SYMBOL("m_data"), sizeof(m), hipMemcpyDeviceToHost);        
       int *dev_out, *dev_n1, *dev_n2, *dev_n3, *dev_n4, *dev_n5, *dev_n6; 
       double *dev_p_arr; 
       int host_out[threads*blocks],host_n1[threads*blocks], host_n2[threads*blocks], host_n3[threads*blocks], host_n4[threads*blocks], host_n5[threads*blocks], host_n6[threads*blocks]; 
       int host_out1[threads*blocks]; 
       double p_arr[6]={0.5,0.5,0.5,0.5,0.5,0.5}; 
       double p[6]; 
       
       hipMalloc( (void**)&dev_n1, sizeof(host_n1));
       hipMalloc( (void**)&dev_n2, sizeof(host_n1));
       hipMalloc( (void**)&dev_n3, sizeof(host_n1));
       hipMalloc( (void**)&dev_n4, sizeof(host_n1));
       hipMalloc( (void**)&dev_n5, sizeof(host_n1));
       hipMalloc( (void**)&dev_n6, sizeof(host_n1));
       hipMalloc( (void**)&dev_out, sizeof(host_n1));
       hipMalloc( (void**)&dev_p_arr, sizeof(p_arr)); 
       
       for(int k=0; k<50; k++) 
       {

       hipMemcpy(dev_p_arr, &p_arr, sizeof(p_arr), hipMemcpyHostToDevice); 
       
       max_cut<<<blocks,threads>>>(time(NULL), dev_p_arr, dev_out, dev_n1, dev_n2, dev_n3, dev_n4, dev_n5, dev_n6);
       
       hipMemcpy(&host_out, dev_out, sizeof(host_n1), hipMemcpyDeviceToHost); 
       hipMemcpy(&host_out1, dev_out, sizeof(host_n1), hipMemcpyDeviceToHost); 
       hipMemcpy(&host_n1, dev_n1, sizeof(host_n1), hipMemcpyDeviceToHost);
       hipMemcpy(&host_n2, dev_n2, sizeof(host_n1), hipMemcpyDeviceToHost);
       hipMemcpy(&host_n3, dev_n3, sizeof(host_n1), hipMemcpyDeviceToHost);
       hipMemcpy(&host_n4, dev_n4, sizeof(host_n1), hipMemcpyDeviceToHost);
       hipMemcpy(&host_n5, dev_n5, sizeof(host_n1), hipMemcpyDeviceToHost);
       hipMemcpy(&host_n6, dev_n6, sizeof(host_n1), hipMemcpyDeviceToHost);
       
       thrust::sort_by_key(host_out, host_out+threads*blocks, host_n1); 
       memcpy(host_out, host_out1, sizeof(host_out)); 
       
       thrust::sort_by_key(host_out, host_out+threads*blocks, host_n2);
       memcpy(host_out, host_out1, sizeof(host_out));
       
       thrust::sort_by_key(host_out, host_out+threads*blocks, host_n3);
       memcpy(host_out, host_out1, sizeof(host_out));
       
       thrust::sort_by_key(host_out, host_out+threads*blocks, host_n4);
       memcpy(host_out, host_out1, sizeof(host_out));
       
       thrust::sort_by_key(host_out, host_out+threads*blocks, host_n5);
       memcpy(host_out, host_out1, sizeof(host_out));
       
       thrust::sort_by_key(host_out, host_out+threads*blocks, host_n6);
       memcpy(host_out, host_out1, sizeof(host_out));
       
       memcpy(p, p_arr, sizeof(p_arr)); 
       memcpy(p_arr, arr_zero, sizeof(arr_zero)); 
	        for(int i = 0.9*threads*blocks; i<threads*blocks; i++)
               {
		       p_arr[0]+= host_n1[i]/double(0.1*threads*blocks); 
		       p_arr[1]+= host_n2[i]/double(0.1*threads*blocks);
		       p_arr[2]+= host_n3[i]/double(0.1*threads*blocks);
		       p_arr[3]+= host_n4[i]/double(0.1*threads*blocks);
		       p_arr[4]+= host_n5[i]/double(0.1*threads*blocks);
                       p_arr[5]+= host_n6[i]/double(0.1*threads*blocks);
	       }
      	     
		err = 0; 
		for(int i=0;i<6;i++)
		     err += pow((p[i]-p_arr[i]),2);
	
		      if(sqrt(err)<1E-4)
		         break; 
		        else
	                cout<<err<<" "; 


		cout<<host_n1[threads*blocks-1]<< " "<<host_n2[threads*blocks-1]<<" "<<host_n3[threads*blocks-1]<<" "<<host_n4[threads*blocks-1]<<" "<<host_n5[threads*blocks-1]<<" "<<host_n6[threads*blocks-1]<<" "<<host_out[threads*blocks-1]<<"\n";
             cout<<p_arr[0]<<" "<<p_arr[1]<<" "<<p_arr[2]<<" "<<p_arr[3]<<" "<<p_arr[4]<<" "<<p_arr[5]<<"\n"; 
	       //cout<<host_data[i]<<" "; 
       }
       return 0;
}
